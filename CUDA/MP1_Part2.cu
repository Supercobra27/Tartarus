#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <assert.h>
#include <random>
#include <chrono>
#include <fstream>
#include <iomanip>

/**
 * @author Ryan Silverberg
 * @student_id: 20342023
*/

#define MEMCHECK(call) if(!call) exit(EXIT_FAILURE);

typedef float* mat_t;

using namespace std;
using namespace std::chrono;

mat_t cpuMatrix;

// One Block, Many Threads
__global__ void matmulGPU(mat_t N, mat_t M, mat_t P, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float Pv = 0;
        for (int k = 0; k < n; ++k) {
            Pv += M[row * n + k] * N[k * n + col];
        }
        P[row * n + col] = Pv;
    }
}

__global__ void matmulGPU_1B(mat_t M, mat_t N, mat_t P, int n) {
    for (int i = 0; i < n * n; i++) {
        P[i] = 0.0f;
    }
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < n; k++) {
                P[i * n + j] += M[i * n + k] * N[k * n + j];
            }
        }
    }
}

void matmulCPU(mat_t M, mat_t N, mat_t P, int n) {
    for (int i = 0; i < n * n; i++) {
        P[i] = 0.0f;
    }
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < n; k++) {
                P[i * n + j] += M[i * n + k] * N[k * n + j];
            }
        }
    }
}

// make this single precision
void initMatrix(mat_t m, int n) {
    srand(374);
    for (int i = 0; i < n * n; i++) {
        m[i] = (float)(rand()) / (float)(rand());
    }
}

void printMatrix(mat_t m, int n) {
    for (int i = 0; i < n * n; i++) {
        printf("%.2f ", m[i]);
        if ((i % n) == n - 1) printf("\n");
    }
}

hipError_t compareMatrices(mat_t CPU, mat_t GPU, int n, float error) {
    for (int i = 0; i < n * n; i++) {
        if (fabs(CPU[i] - GPU[i]) > error) {
            printf("Mismatch at element %d: CPU=%f, GPU=%f\n", i, CPU[i], GPU[i]);
            return hipErrorInvalidValue;
        }
    }
    return hipSuccess;
}

#define TOLERANCE 1e-6

typedef struct _transfer {
    float toCPU;
    float toGPU;
}transfer;

transfer transfer_time(int n) {

    size_t size = n * n * sizeof(float); // Size of matrix in bytes
    hipEvent_t toGPUstart, toGPUend; // Host -> Device
    hipEvent_t toCPUstart, toCPUend; // Device -> Host

    mat_t M, N; // CPU
    mat_t gM, gN; // GPU

    hipEventCreate(&toGPUstart);
    hipEventCreate(&toGPUend);
    hipEventCreate(&toCPUstart);
    hipEventCreate(&toCPUend);

    hipDeviceSynchronize();

    hipHostMalloc((void**)&M, size);
    hipHostMalloc((void**)&N, size);
    hipMalloc((void**)&gM, size);
    hipMalloc((void**)&gN, size);

    initMatrix(M, n);
    initMatrix(N, n);

    transfer ret;

    // Host --> Device

    hipEventRecord(toGPUstart, 0);
    hipMemcpy(gM, M, size, hipMemcpyHostToDevice);
    hipMemcpy(gN, N, size, hipMemcpyHostToDevice);
    hipEventRecord(toGPUend, 0);
    hipEventSynchronize(toGPUend);
    hipEventElapsedTime(&ret.toGPU, toGPUstart, toGPUend);

    // Device --> Host

    hipEventRecord(toCPUstart, 0);
    hipMemcpy(M, gM, size, hipMemcpyDeviceToHost);
    hipMemcpy(N, gN, size, hipMemcpyDeviceToHost);
    hipEventRecord(toCPUend, 0);
    hipEventSynchronize(toCPUend);
    hipEventElapsedTime(&ret.toCPU, toCPUstart, toCPUend);


    hipFree(gM);
    hipFree(gN);
    hipHostFree(M);
    hipHostFree(N);

    hipEventDestroy(toGPUstart);
    hipEventDestroy(toGPUend);
    hipEventDestroy(toCPUstart);
    hipEventDestroy(toCPUend);

    hipDeviceReset();

    return ret;
}

float single_thread_time(int n) {
    float ret;
    size_t size = n * n * sizeof(float); // Size of matrix in bytes
    hipEvent_t toGPUstart, toGPUend; // for Block Calculations

    mat_t M, N, P; // CPU
    mat_t gM, gN, gP; // GPU

    hipEventCreate(&toGPUstart);
    hipEventCreate(&toGPUend);

    hipDeviceSynchronize();

    hipHostMalloc((void**)&M, size);
    hipHostMalloc((void**)&N, size);
    hipHostMalloc((void**)&P, size);
    hipMalloc((void**)&gM, size);
    hipMalloc((void**)&gN, size);
    hipMalloc((void**)&gP, size);

    initMatrix(M, n);
    initMatrix(N, n);

    // Host --> Device

    hipMemcpy(gM, M, size, hipMemcpyHostToDevice);
    hipMemcpy(gN, N, size, hipMemcpyHostToDevice);

    int numBlocks = n / n;
    if (n % n) numBlocks++;

    dim3 dimGrid(1);
    dim3 dimBlock(1);

    hipEventRecord(toGPUstart, 0);
    matmulGPU_1B << <dimGrid, dimBlock >> > (gM, gN, gP, n);
    hipEventRecord(toGPUend, 0);
    hipEventSynchronize(toGPUend);
    hipEventElapsedTime(&ret, toGPUstart, toGPUend);

    // Device --> Host

    hipMemcpy(P, gP, size, hipMemcpyDeviceToHost);

    if (!compareMatrices(cpuMatrix, P, n, TOLERANCE)) printf("Test 1 PASSED\n");

    hipFree(gM);
    hipFree(gN);
    hipFree(gP);
    hipHostFree(M);
    hipHostFree(N);
    hipHostFree(P);

    hipEventDestroy(toGPUstart);
    hipEventDestroy(toGPUend);

    hipDeviceReset();

    return ret;
}

int cpu_matmul_time(int n) {

    size_t size = n * n * sizeof(float);
    cpuMatrix = new float[n * n];

    mat_t M, N, P;

    hipHostMalloc((void**)&M, size);
    hipHostMalloc((void**)&N, size);
    hipHostMalloc((void**)&P, size);

    initMatrix(M, n);
    initMatrix(N, n);


    auto cpu_start = high_resolution_clock::now();
    matmulCPU(M, N, P, n);
    auto cpu_end = high_resolution_clock::now();

    auto duration = duration_cast<milliseconds>(cpu_end - cpu_start);

    cpuMatrix = P;

    hipHostFree(M);
    hipHostFree(N);
    hipHostFree(P);

    return duration.count();
}

float block_change_time(int n, int b) {

    float ret;
    size_t size = n * n * sizeof(float); // Size of matrix in bytes
    hipEvent_t toGPUstart, toGPUend; // for Block Calculations

    mat_t M, N, P; // CPU
    mat_t gM, gN, gP; // GPU

    hipEventCreate(&toGPUstart);
    hipEventCreate(&toGPUend);

    hipDeviceSynchronize();

    hipHostMalloc((void**)&M, size);
    hipHostMalloc((void**)&N, size);
    hipHostMalloc((void**)&P, size);
    hipMalloc((void**)&gM, size);
    hipMalloc((void**)&gN, size);
    hipMalloc((void**)&gP, size);

    initMatrix(M, n);
    initMatrix(N, n);

    // Host --> Device

    hipMemcpy(gM, M, size, hipMemcpyHostToDevice);
    hipMemcpy(gN, N, size, hipMemcpyHostToDevice);

    int numBlocks = n / b;
    if (n % b) numBlocks++;

    dim3 dimGrid(numBlocks, numBlocks);
    dim3 dimBlock(b, b);

    hipEventRecord(toGPUstart, 0);
    matmulGPU << <dimGrid, dimBlock >> > (gM, gN, gP, n);
    hipEventRecord(toGPUend, 0);
    hipEventSynchronize(toGPUend);
    hipEventElapsedTime(&ret, toGPUstart, toGPUend);

    // Device --> Host

    hipMemcpy(P, gP, size, hipMemcpyDeviceToHost);

    hipFree(gM);
    hipFree(gN);
    hipFree(gP);
    hipHostFree(M);
    hipHostFree(N);
    hipHostFree(P);

    hipEventDestroy(toGPUstart);
    hipEventDestroy(toGPUend);

    hipDeviceReset();

    return ret;

}


typedef struct _timing {
    int width;
    float toCPUtime;
    float toGPUtime;
    float blkGPUtime[5];
    float singleGPUtime;
    int mulCPUtime;

}timing;

timing run_mp(int n) {
    // Part 1: Find Transfer Times
    timing trial_out;

    trial_out.width = n;

    transfer t = transfer_time(n);

    printf("Transfer to GPU took %.2f ms\n", t.toGPU);
    printf("Transfer to CPU took %.2f ms\n", t.toCPU);

    trial_out.toGPUtime = t.toGPU;
    trial_out.toCPUtime = t.toCPU;

    // Part 2: Find Single Threaded Times
    if (!(n == 2048 || n == 4096)) {
        trial_out.mulCPUtime = cpu_matmul_time(n);
        trial_out.singleGPUtime = single_thread_time(n);

        printf("GPU-single-threaded took %.2f ms\n", trial_out.singleGPUtime);
        printf("CPU matrix calculation was %d ms\n", trial_out.mulCPUtime);
    }

    // Part 3: Find Block Execution Times
    int blksiz[5] = { 2, 4, 8, 16, 32 };
    for (int b = 0; b < 5; b++) {
        trial_out.blkGPUtime[b] = block_change_time(n, blksiz[b]);
        printf("Calculation with Block Size %d took %.2f ms\n", blksiz[b], trial_out.blkGPUtime[b]);
    }

    return trial_out;

}
#define NUM_TRIALS 5
int main() {
    int widths[5] = { 256, 512, 1024, 2048, 4096 };
    std::ofstream csvfile("data.csv");
    cpuMatrix = new float;

    timing times;
    for (int t = 0; t < NUM_TRIALS; t++) {
        for (int w = 0; w < 5; w++) {
            times = run_mp(widths[w]);
            csvfile << times.width << ',' << times.toGPUtime << ',' << times.toCPUtime << ',' << times.mulCPUtime << ',' << times.singleGPUtime <<
                times.blkGPUtime[0] << ',' <<
                times.blkGPUtime[1] << ',' <<
                times.blkGPUtime[2] << ',' <<
                times.blkGPUtime[3] << ',' <<
                times.blkGPUtime[4] << endl;
        }
    }

    delete cpuMatrix;
    csvfile.close();

    return 0;
}
